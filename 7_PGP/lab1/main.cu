
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define CSC(call)  \
do { \
	hipError_t state = call; \
	if (state != hipSuccess) { \
		fprintf(stderr, "ERROR: %s:%d. Message: %s\n", __FILE__,__LINE__,hipGetErrorString(state)); \
		exit(0); \
	} \
} while (0); \

__global__ void kernel(double* v1, double* v2, double* v3, long long n) {
	long long i = blockDim.x * blockIdx.x + threadIdx.x;
	long long offset = blockDim.x * gridDim.x;

	while (i < n) {
		if (v1[i] < v2[i]) {
			v3[i] = v1[i];
		}
		else {
			v3[i] = v2[i];
		}
		i = i + offset;
	}
}

int main() {
	long long n;
	scanf("%lld", &n);

	double* v1 = (double*)malloc(n * sizeof(double));
	double* v2 = (double*)malloc(n * sizeof(double));

	for(long long i = 0; i < n; i++) {
		scanf("%lf", &v1[i]);
	}
	for(long long i = 0; i < n; i++) {
			scanf("%lf", &v2[i]);
	}

	double* dev_v1;
	double* dev_v2;
	double* dev_v3;
	CSC(hipMalloc(&dev_v1, sizeof(double) * n));
	CSC(hipMalloc(&dev_v2, sizeof(double) * n));
	CSC(hipMalloc(&dev_v3, sizeof(double) * n));

	CSC(hipMemcpy(dev_v1, v1, sizeof(double) * n, hipMemcpyHostToDevice));
	CSC(hipMemcpy(dev_v2, v2, sizeof(double) * n, hipMemcpyHostToDevice));

	kernel<<<256,256>>>(dev_v1, dev_v2, dev_v3, n);

	CSC(hipMemcpy(v1, dev_v3, sizeof(double) * n, hipMemcpyDeviceToHost));
	CSC(hipFree(dev_v1));
	CSC(hipFree(dev_v2));
	CSC(hipFree(dev_v3));

	for (long long i = 0; i < n; i++) {
		printf("%.10lf ", v1[i]);
	}
	printf("\n");
	free(v1);
	free(v2);
	return 0;
}
