
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define CSC(call)  \
do { \
	hipError_t state = call; \
	if (state != hipSuccess) { \
		fprintf(stderr, "ERROR: %s:%d. Message: %s\n", __FILE__,__LINE__,hipGetErrorString(state)); \
		exit(0); \
	} \
} while (0); \

__global__ void kernel(double* v1, double* v2, double* v3, long long n) {
	long long i = blockDim.x * blockIdx.x + threadIdx.x;
	long long offset = blockDim.x * gridDim.x;

	while (i < n) {
		if (v1[i] < v2[i]) {
			v3[i] = v1[i];
		}
		else {
			v3[i] = v2[i];
		}
		i = i + offset;
	}
}

int main() {
	long long n;
	scanf("%lld", &n);

	double* v1 = (double*)malloc(n * sizeof(double));
	double* v2 = (double*)malloc(n * sizeof(double));

	for(long long i = 0; i < n; i++) {
		scanf("%lf", &v1[i]);
	}
	for(long long i = 0; i < n; i++) {
			scanf("%lf", &v2[i]);
	}

	double* dev_v1;
	double* dev_v2;
	double* dev_v3;
	CSC(hipMalloc(&dev_v1, sizeof(double) * n));
	CSC(hipMalloc(&dev_v2, sizeof(double) * n));
	CSC(hipMalloc(&dev_v3, sizeof(double) * n));

	CSC(hipMemcpy(dev_v1, v1, sizeof(double) * n, hipMemcpyHostToDevice));
	CSC(hipMemcpy(dev_v2, v2, sizeof(double) * n, hipMemcpyHostToDevice));
	
	hipEvent_t start, end;
	for (int block = 1; block <= 1024; block *= 2) {
		for (int threads = 32; threads <= 1024; threads *= 2) {
			CSC(hipEventCreate(&start));
			CSC(hipEventCreate(&end));
			CSC(hipEventRecord(start));
	
			kernel<<<block,threads>>>(dev_v1, dev_v2, dev_v3, n);
	
			CSC(hipEventRecord(end));
			CSC(hipEventSynchronize(end));
			float t;
			CSC(hipEventElapsedTime(&t, start, end));
			CSC(hipEventDestroy(start));
			CSC(hipEventDestroy(end));

			printf("kernel = <<<%d, %d>>>, time = %f\n", block, threads, t);
		}
	}

	CSC(hipMemcpy(v1, dev_v3, sizeof(double) * n, hipMemcpyDeviceToHost));
	CSC(hipFree(dev_v1));
	CSC(hipFree(dev_v2));
	CSC(hipFree(dev_v3));

	for (long long i = 0; i < n; i++) {
		printf("%.10lf ", v1[i]);
	}
	printf("\n");
	free(v1);
	free(v2);
	return 0;
}
