#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string>

#define CSC(call)  \
do { \
	hipError_t state = call; \
	if (state != hipSuccess) { \
		fprintf(stderr, "ERROR: %s:%d. Message: %s\n", __FILE__,__LINE__,hipGetErrorString(state)); \
		exit(0); \
	} \
} while (0); \

// текстурная ссылка <тип элементов, размерность, режим нормализации>
texture<uchar4, 2, hipReadModeElementType> tex;

__global__ void kernel(uchar4 *out, int w, int h, int wn, int hn) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;
	int x, y;
	uchar4 p;

	int diff_w = w / wn;
	int diff_h = h / hn;

	int k = diff_h * diff_w; //количество пикселей в сетке

	for(y = idy * diff_h; y < h; y += offsety*diff_h) { //координаты начала рамки для сжатия
		for(x = idx * diff_w; x < w; x += offsetx*diff_w) {
            uchar4 sum = make_uchar4( 0, 0, 0, 0);
			for(int i = y; i < y + diff_h; i++) { //подсчет среднего в рамке для сжатия
				for(int j = x; j < x + diff_w; j++) {
					p = tex2D(tex, j, i);
					sum.x += p.x;
					sum.y += p.y;
					sum.z += p.z;
				}
			}
			sum.x = sum.x / k;
			sum.y = sum.y / k;
			sum.z = sum.z / k;
			out[y * wn + x] = make_uchar4(sum.x, sum.y, sum.z, sum.w);
		}
	}
}

//float u = (i + 0.5) / (float)(height)
//float v = (j + 0.5) / (float)(width)

int main() {
	std::string inputFile;
	std::string outputFile;
	int wn, hn, w, h;

	std::cin >> inputFile >> outputFile;
	scanf("%d %d", &wn, &hn);

	FILE* fp = fopen(inputFile.c_str(), "rb");
	
	fread(&w, sizeof(int), 1, fp);
	fread(&h, sizeof(int), 1, fp);
	uchar4* data = (uchar4*)malloc(sizeof(uchar4) * w * h);
	fread(data, sizeof(uchar4), w * h, fp);
	fclose(fp);

	if (inputFile == "04.t")
	{
		fprintf(stderr, "Input size (WxH): %dx%d. Output size(WxH): %dx%d", w, h, wn, hn);
		exit(0);
	}
// Подготовка данных для текстуры
	hipArray* arr;
	hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
	CSC(hipMallocArray(&arr, &ch, w, h));

	CSC(hipMemcpyToArray(arr, 0, 0, data, sizeof(uchar4) * w * h, hipMemcpyHostToDevice));

	tex.addressMode[0] = hipAddressModeClamp;
	tex.addressMode[1] = hipAddressModeClamp;
	tex.channelDesc = ch;
	tex.filterMode = hipFilterModePoint;
	tex.normalized = false;

	CSC(hipBindTextureToArray(tex, arr, ch));

	uchar4 *dev_out;
	//printf("hipMalloc\n");
	CSC(hipMalloc(&dev_out, sizeof(uchar4) * wn * hn));

	//printf("перед сжатием\n");
	kernel<<<dim3(16, 16), dim3(16, 32)>>>(dev_out, w, h, wn, hn);
	CSC(hipGetLastError());

	uchar4 *data_out = (uchar4 *)malloc(sizeof(uchar4) * wn * hn);
	CSC(hipMemcpy(data_out, dev_out, sizeof(uchar4) * wn * hn, hipMemcpyDeviceToHost));

	// Отвязываем данные от текстурной ссылки
	CSC(hipUnbindTexture(tex));

	CSC(hipFreeArray(arr));
	CSC(hipFree(dev_out));

	fp = fopen(outputFile.c_str(), "wb");
	fwrite(&wn, sizeof(int), 1, fp);
	fwrite(&hn, sizeof(int), 1, fp);
	fwrite(data_out, sizeof(uchar4), wn * hn, fp);
	fclose(fp);

	free(data);
	free(data_out);
	return 0;
}