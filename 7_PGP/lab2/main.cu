#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#define CSC(call)  \
do { \
	hipError_t state = call; \
	if (state != hipSuccess) { \
		fprintf(stderr, "ERROR: %s:%d. Message: %s\n", __FILE__,__LINE__,hipGetErrorString(state)); \
		exit(0); \
	} \
} while (0); \

// текстурная ссылка <тип элементов, размерность, режим нормализации>
texture<uchar4, 2, hipReadModeElementType> tex;

__global__ void kernel(uchar4 *out, int w, int h, int wn, int hn) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;
	int x, y;
	uchar4 p;

	int diff_w = w / wn;
	int diff_h = h / hn;

	uchar4 sum = make_uchar4( 0, 0, 0, 0);
	int k = 0; //количество пикселей в сетке

	for(y = idy; y < h; y += offsety) { //координаты начала рамки для сжатия
		for(x = idx; x < w; x += offsetx) {
			for(int i = y; i <= y + diff_h; i++) { //подсчет среднего в рамке для сжатия
				for(int j = x; j <= x + diff_w; j++) {
					p = tex2D(tex, x, y);
					sum.x += ~p.x;
					sum.y += ~p.y;
					sum.z += ~p.z;
					sum.w += p.w;
					k += 1;
				}
			}
			sum.x = sum.x / k;
			sum.y = sum.y / k;
			sum.z = sum.z / k;
			sum.w = sum.w / k;
			out[y * wn + x] = make_uchar4(~sum.x, ~sum.y, ~sum.z, sum.w);
		}
	}
}

//float u = (i + 0.5) / (float)(height)
//float v = (j + 0.5) / (float)(width)

int main() {
	char path_in[100];
	char path_out[100];
	int w, h, wn, hn;

    fgets(path_in, sizeof(path_in), stdin);
	fgets(path_out, sizeof(path_out), stdin);
	scanf("%d", &wn);
	scanf("%d", &hn);

	FILE *fp = fopen(path_in, "rb"); //сохраняю данные картинки
	fread(&w, sizeof(int), 1, fp);
	fread(&h, sizeof(int), 1, fp);
	uchar4 *data = (uchar4 *)malloc(sizeof(uchar4) * w * h);
	fread(data, sizeof(uchar4), w * h, fp);
	fclose(fp);

// Подготовка данных для текстуры
	hipArray *arr;
	hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
	CSC(hipMallocArray(&arr, &ch, w, h));

	CSC(hipMemcpyToArray(arr, 0, 0, data, sizeof(uchar4) * w * h, hipMemcpyHostToDevice));

	// Подготовка текстурной ссылки, настройка интерфейса работы с данными
	tex.addressMode[0] = hipAddressModeClamp;	// Политика обработки выхода за границы по каждому измерению
	tex.addressMode[1] = hipAddressModeClamp;
	tex.channelDesc = ch;
	tex.filterMode = hipFilterModePoint;		// Без интерполяции при обращении по дробным координатам
	tex.normalized = false;						// Режим нормализации координат: без нормализации

	// Связываем интерфейс с данными
	CSC(hipBindTextureToArray(tex, arr, ch));

	uchar4 *dev_out;
	CSC(hipMalloc(&dev_out, sizeof(uchar4) * wn * hn));

	kernel<<<dim3(16, 16), dim3(16, 32)>>>(dev_out, w, h, wn, hn);
	CSC(hipGetLastError());

	uchar4 *data_out = (uchar4 *)malloc(sizeof(uchar4) * wn * hn);
	CSC(hipMemcpy(data_out, dev_out, sizeof(uchar4) * w * h, hipMemcpyDeviceToHost));

	// Отвязываем данные от текстурной ссылки
	CSC(hipUnbindTexture(tex));

	CSC(hipFreeArray(arr));
	CSC(hipFree(dev_out));

	fp = fopen(path_out, "wb");
	fwrite(&wn, sizeof(int), 1, fp);
	fwrite(&hn, sizeof(int), 1, fp);
	fwrite(data_out, sizeof(uchar4), wn * hn, fp);
	fclose(fp);

	free(data);
	free(data_out);
	return 0;
}
