
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <vector>
#include <string>

#define CSC(call)  \
do { \
	hipError_t state = call; \
	if (state != hipSuccess) { \
		fprintf(stderr, "ERROR: %s:%d. Message: %s\n", __FILE__,__LINE__,hipGetErrorString(state)); \
		exit(0); \
	} \
} while (0); \

typedef struct pair {
    double x;
    double y;
} pair;

__host__ double dist(double x1, double y1, double z1, double x2, double y2, double z2) {
    return sqrt((x1-x2)*(x1-x2) + (y1-y2)*(y1-y2) + (z1-z2)*(z1-z2));
}

__device__ double dist1(double x1, double y1, double z1, double x2, double y2, double z2) {
    return sqrt((x1-x2)*(x1-x2) + (y1-y2)*(y1-y2) + (z1-z2)*(z1-z2));
}

__constant__ pair u[32];

__global__ void kernel(uchar4 *data, int w, int h, int n) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;
	int x, y;
	double mas[32];
    for (y = idy; y < h; y += offsety) {
        for(x = idx; x < w; x += offsetx) {
            double4 p = make_double4(data[y * w + x].x, data[y * w + x].y, data[y * w + x].z, 0);//int4
            for (int j = 0; j < n; j++) {
                 double4 c = make_double4(data[int(u[j].y) * w + int(u[j].x)].x, data[int(u[j].y) * w + int(u[j].x)].y, data[int(u[j].y) * w + int(u[j].x)].z, 0); //int4
                 mas[j] = dist1(c.x, c.y, c.z, p.x, p.y, p.z); //заполняем массив расстояний            
            }
            //поиск минимума
            double MIN = mas[0];
            int min_i = 0;
            for (int j = 0; j < n; j++) {
                if (mas[j] < MIN) {
                    MIN = mas[j];
                    min_i = j; 
                }
            }
            p.w = min_i;
            data[y*w+x] = make_uchar4(p.x, p.y, p.z, p.w);
        }
    }         
}

int main() {
	std::string inputFile;
	std::string outputFile;
	int n, w, h;
    double x, y;

	std::cin >> inputFile >> outputFile;
	scanf("%d", &n);
    pair centre[n];
    pair centre_new[n];
    for(int i = 0; i < n; i++) {
        scanf("%lf %lf", &x, &y);
        centre[i].x = x;
        centre[i].y = y;
    }

	FILE* fp = fopen(inputFile.c_str(), "rb");
	
	fread(&w, sizeof(int), 1, fp);
	fread(&h, sizeof(int), 1, fp);
	uchar4* data = (uchar4*)malloc(sizeof(uchar4) * w * h);
	fread(data, sizeof(uchar4), w * h, fp);
	fclose(fp);

    uchar4 *dev_data;
    CSC(hipMalloc(&dev_data, sizeof(uchar4) * h * w));
    CSC(hipMemcpy(dev_data, data, sizeof(uchar4) * h * w, hipMemcpyHostToDevice));

    bool flag = true;
    while(flag) {
        CSC(hipMemcpyToSymbol(HIP_SYMBOL(u), centre, sizeof(pair) * n, 0, hipMemcpyHostToDevice));
        kernel<<<dim3(16, 16), dim3(16, 16)>>>(dev_data, w, h, n); 
        CSC(hipMemcpy(data, dev_data, sizeof(uchar4) * h * w, hipMemcpyDeviceToHost));
       
        //обновление центра кластеров
        long long int k[n];
        long long int sum[n][2];
        for(int y = 0; y < n; y++) {
            k[y] = 0;
            sum[y][0] = 0;
            sum[y][1] = 0;
        }
        int4 t;
        for(int y = 0; y < h; y++) {
            for(int x = 0; x < w; x++) {
                t = make_int4(data[y*w+x].x, data[y*w+x].y, data[y*w+x].z, data[y*w+x].w);
                k[t.w] += 1;
                sum[t.w][0] = y;
                sum[t.w][1] = x;
            }
        }
        for(int r = 0; r < n; r++) {
            centre_new[r].x = sum[r][1] / k[r];
            centre_new[r].y = sum[r][0] / k[r];
        }
        
        //условие прекращения
        for(int i = 0; i < n; i++) {
            if (centre[i].x == centre_new[i].x && centre[i].y == centre_new[i].y) {
                //проверяем дальше
            }
            else {
                flag = false; //были не одинаковые
            }    
        }
        if (flag == false) {
            flag = true; //продолжаем обработку
            for(int i = 0; i < n; i++) { 
                centre[i].x = centre_new[i].x;
                centre[i].y = centre_new[i].y;
            }
        }
        else {
            flag = false; //заканчиваем, выход из цикла
        }
    }

    fp = fopen(outputFile.c_str(), "wb");
	fwrite(&w, sizeof(int), 1, fp);
	fwrite(&h, sizeof(int), 1, fp);
	fwrite(data, sizeof(uchar4), w * h, fp);
	fclose(fp);

	free(data);
    CSC(hipFree(dev_data));
	return 0;
}