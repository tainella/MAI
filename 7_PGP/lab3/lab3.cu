#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <vector>
#include <string>

#define CSC(call)  \
do { \
	hipError_t state = call; \
	if (state != hipSuccess) { \
		fprintf(stderr, "ERROR: %s:%d. Message: %s\n", __FILE__,__LINE__,hipGetErrorString(state)); \
		exit(0); \
	} \
} while (0); \

typedef struct pair {
    double x;
    double y;
} pair;

__host__ double dist(double x1, double y1, double z1, double x2, double y2, double z2) {
    return sqrt((x1-x2)*(x1-x2) + (y1-y2)*(y1-y2) + (z1-z2)*(z1-z2));
}

__constant__ pair u[];

__global__ void kernel(uchar4 *data, int w, int h, int n) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;
	int x, y;
	
    for (y = idy; y < h; y += offsety) {
        for(x = idx; k < w; x < w; x += offsetx) {
            double mas[n];
            double4 p = data[y * w + x]; //int4
            for (int j = 0; j < n; j++) {
                double4 c = data[u[j].y * w + u[j].x];
                mas[j] = dist(c.x, c.y, c.z, p.x, p.y, p.z); //заполняем массив расстояний            
            }
            //поиск минимума
            double MIN = mas[0];
            int min_i = 0;
            for (int j = 0; j < n; j++) {
                if (mas[j] < MIN) {
                    MIN = mas[j];
                    min_i = j; 
                }
            }
            p.w = min_i;
            data[y*w+x] = p;
        }
    }         
}

int main() {
	std::string inputFile;
	std::string outputFile;
	int n, x, y;

	std::cin >> inputFile >> outputFile;
	scanf("%d", &n);
    pair centre[n];
    pair centre_new[n];
    for(int i = 0; i < n; i++) {
        scanf("%lf %lf", &x, &y);
        centre[i].x = x;
        centre[i].y = y;
    }

	FILE* fp = fopen(inputFile.c_str(), "rb");
	
	fread(&w, sizeof(int), 1, fp);
	fread(&h, sizeof(int), 1, fp);
	uchar4* data = (uchar4*)malloc(sizeof(uchar4) * w * h);
	fread(data, sizeof(uchar4), w * h, fp);
	fclose(fp);

    uchar4 *dev_data;
    CSC(hipMalloc(&dev_data, sizeof(uchar4) * h * w));
    CSC(hipMemcpy(dev_data, data, sizeof(uchar4) * h * w, hipMemcpyHostToDevice));

    bool flag = true;
    while(flag) {
        CSC(hipMemcpyToSymbol(HIP_SYMBOL(u), centre, sizeof(pair) * количество, 0, hipMemcpyHostToDevice));
        kernel<<<dim3(16, 16), dim3(16, 16)>>>(dev_data, w, h, n); 
        CSC(hipMemcpy(data, dev_data, sizeof(uchar4) * h * w, hipMemcpyDeviceToHost));
       
        //обновление центра кластеров
        long long int k[n] = 0;
        long long int sum[n][2];
        for(int y = 0; y < n; y++) {
            sum[y][0] = 0;
            sum[y][1] = 0;
        }
        int4 t;
        for(int y = 0; y < h; y++) {
            for(int x = 0; x < w; x++) {
                t = data[y*w+x];
                k[t.w] += 1;
                sum[t.w][0] = y;
                sum[t.w][1] = x;
            }
        }
        for(int r = 0; r < n; r++) {
            centre_new[r].x = sum[r][1] / k[r];
            centre_new[r].y = sum[r][0] / k[r];
        }
        
        //условие прекращения
        for(int i = 0; i < n; i++) {
            if (centre[i].x == centre_new[i].x && centre[i].y == centre_new[i].y) {
                //проверяем дальше
            }
            else {
                flag = false; //были не одинаковые
            }    
        }
        if (flag == false) {
            flag = true; //продолжаем обработку
            for(int i = 0; i < n; i++) { 
                centre[i].x = centre_new[i].x;
                centre[i].y = centre_new[i].y;
            }
        }
        else {
            flag = false; //заканчиваем, выход из цикла
        }
    }

    fp = fopen(outputFile.c_str(), "wb");
	fwrite(&w, sizeof(int), 1, fp);
	fwrite(&h, sizeof(int), 1, fp);
	fwrite(data, sizeof(uchar4), w * h, fp);
	fclose(fp);

	free(data);
	free(data_out);
	return 0;
}

