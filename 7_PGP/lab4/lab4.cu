#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <vector>
#include <string>

#include <thrust/extrema.h>
#include <thrust/device_vector.h>

#define CSC(call)  \
do { \
	hipError_t state = call; \
	if (state != hipSuccess) { \
		fprintf(stderr, "ERROR: %s:%d. Message: %s\n", __FILE__,__LINE__,hipGetErrorString(state)); \
		exit(0); \
	} \
} while (0); \

struct action {
    __host__ __device__ bool operator() (double x, double y) {
        return std::fabs(x) < std::fabs(y);
    }
};

__global__ void swap(double* a, int k, int y, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int offsetx = blockDim.x * gridDim.x;
    double w;
    for(int i = idx; i < n+1; i += offsetx) {
        w = a[i * n + k];
        a[i * n + k] = a[i * n + y];
        a[i * n + y] = w;
    }
}

__global__ void change(double *data, int n, int m)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offsetx = blockDim.x * gridDim.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
	int offsety = blockDim.y * gridDim.y;

    for (int j = idx+ m +1; j < n; j += offsetx) {
            double d = data[m*n+j] / data[m*n+m];
            for (int i = m + idy; i < n + 1; i+=offsety) { //для приписанной матрицы
                data[i*n+j] = data[i*n+j] - data[i*n+m] * d;
            }
     }
}

int main() {
    int n;
    std::cin >> n;
    double* a = (double*)malloc(sizeof(double) * n * (n+1));
    double* x = (double*)malloc(sizeof(double) * n);
    double p;
    //ввод
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            std::cin >> p;
            a[j*n + i] = p;
        }
    }

    for (int i = 0; i < n; i++) {
            std::cin >> p;
            int s = n*n + i;
            a[s] = p;
    }


    //память куды
    double* data;
    CSC(hipMalloc(&data, sizeof(double) * n * (n+1)));
    CSC(hipMemcpy(data, a, sizeof(double) * n * (n+1), hipMemcpyHostToDevice));

    action comparator;

    thrust::device_ptr<double> i_ptr, i_max_ptr;

    double* dop = (double*)malloc(sizeof(double));
    
    for(int m = 0; m < n-1; m++) {
        int i_max = m;
        i_ptr = thrust::device_pointer_cast(data + m * n);
        i_max_ptr = thrust::max_element(i_ptr + m, i_ptr + n, comparator);
        i_max = i_max_ptr - i_ptr;
        if(m != i_max) {
            swap<<<256, 256>>>(data, m, i_max, n);
        }
        CSC(hipMemcpy(dop, data+m*n+m, sizeof(double), hipMemcpyDeviceToHost));
        if (*dop != 0) {
            change<<<dim3(32, 32), dim3(32, 32)>>>(data, n, m);
        }
    }

    CSC(hipMemcpy(a, data, sizeof(double) * n * (n+1), hipMemcpyDeviceToHost));
    
    x[n - 1] = a[(n+1)*n - 1] / a[(n+1)*n - n - 1];

    for(int k = n-1; k >= 0; k--) {
        double d = 0;
        for (int j = k + 1; j < n; j++) {
            d = d + a[j*n + k] * x[j];
        }
        x[k] = (a[n*n + k] - d) / a[k*n + k];
    }
    std::cout.precision(10);
    std::cout.setf(std::ios::scientific);

    for (int i = 0; i < n; i++) {
          std::cout << x[i] << " ";

    }
    CSC(hipFree(data));
    free(a);
    free(x);
    return 0;
}