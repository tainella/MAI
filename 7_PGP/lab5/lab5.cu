#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <vector>
#include <string>

#include <thrust/extrema.h>
#include <thrust/device_vector.h>

#define THREAD_NUM 1024

#define CSC(call)  \
do { \
	hipError_t state = call; \
	if (state != hipSuccess) { \
		fprintf(stderr, "ERROR: %s:%d. Message: %s\n", __FILE__,__LINE__,hipGetErrorString(state)); \
		exit(0); \
	} \
} while (0); \

__device__ int what_pow(int n) {
    int k = 0;
    while (n > 1) {
        n = n / 2;
        k++;
    }
    return k;
}

__host__ int what_pow_h(int n) {
    int k = 0;
    while (n > 1) {
        n = n / 2;
        k++;
    }
    return k;
}

__device__  int no_conflict_offset(int n, int banks) {
    return ((n) >> banks + (n) >> (2 * what_pow(banks)));
}

__global__ void prescan(int* d_out, const int* d_in, int blocks) {
    int sum;
    int max_block_size = 256; //128
    __shared__ int temp[256];

    int idx = threadIdx.x;
    int ai = idx;

    temp[idx] = 0;
    temp[idx + blockDim.x] = 0;
    
    __syncthreads();

    temp[ai] = d_in[ai];
    temp[128 + ai] = d_in[128 + ai];
    
    int offset = 1;
    for (int d = max_block_size >> 1; d > 0; d >>= 1)
    {
        __syncthreads();

        if (idx < d)
        {
            int ai = offset * ((idx << 1) + 1) - 1;
            int bi = offset * ((idx << 1) + 2) - 1;
            temp[bi] += temp[ai];
        }
        offset <<= 1;
    }
    if (idx == 0) { 
        sum = temp[255];
        //printf("sum:%d\n", sum);
        temp[255] = 0;
    }
    //обратный ход
    for (int d = 1; d < max_block_size; d <<= 1) {
        offset >>= 1;
        __syncthreads();

        if (idx < d)
        {
            int i1 = offset * ((idx << 1) + 1) - 1;
            int i2 = offset * ((idx << 1) + 2) - 1;

            int t = temp[i1];
            temp[i1] = temp[i2];
            temp[i2] += t;
        }
    }
    __syncthreads();
    if (idx == 0) { //переделать из исключающего в включающий
        for (int j = 1; j < 256; j++) {
            d_out[j - 1] = temp[j];
        }
         //printf("sum2: %d\n", sum);
         d_out[255] = sum;
         /*for (int j = 0; j < 256; j++) {
            printf("%d ", temp[j]);
         }
         printf("\n|||||||||\n");*/
    }
}

__global__ void kernel(int* pref, unsigned char* out){
    int idx = blockDim.x * blockIdx.x +  threadIdx.x;
    int step = blockDim.x * gridDim.x;

    for(int tid = idx; tid < 256; tid += step){
        int low = tid ? pref[tid-1] : 0;

        for(int i = pref[tid] - 1; i >= low; --i){
            out[i] = tid;
        }
    }
}

__global__ void hist(unsigned char* array, int n, int* out) {
    __shared__ int temp[257];
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offsetx = blockDim.x * gridDim.x;

    for(int i = idx; i < n; i += offsetx) {
        atomicAdd(temp + array[i], 1);
    }
    __syncthreads();
    if (idx == 0)
    for(int i = 0; i < 257; i++) {
        atomicAdd(out + i, *(temp + i));
    }
}

int main() {
    int n;
    //std::cin >> n;
    fread(&n, sizeof(int), 1, stdin);
    unsigned char* array = (unsigned char*)malloc(sizeof(unsigned char) * n); //uchar
    if (fread(array, sizeof(unsigned char), n, stdin) != n) {
        std::cout << "не считал\n";
    }
    
    /*for (int i = 0; i < n; i++) {
       // std::cin >> array[i];
       array[i] = 255;
    }
    for (int i = 0; i < 10 && i < n; i++) {
        //printf("%02X ", array[i]);
        std::cerr << array[i] << " ";
    }*/
    unsigned char* gpu_array;
    CSC(hipMalloc(&gpu_array, sizeof(unsigned char) * n));
    CSC(hipMemcpy(gpu_array, array, sizeof(unsigned char) * n, hipMemcpyHostToDevice));
    
    int* counts = (int*)malloc(sizeof(int) * 256);
    for (int i = 0; i < 256; i++) {
        counts[i] = 0;
    }
    int* gpu_counts;
    CSC(hipMalloc(&gpu_counts, sizeof(int) * 256));
    CSC(hipMemcpy(gpu_counts, counts, sizeof(int) * 256, hipMemcpyHostToDevice));

    int* gpu_pref;
    CSC(hipMalloc(&gpu_pref, sizeof(int) * 256));
    CSC(hipMemset(gpu_pref, 0, sizeof(int) * 256));

    hist<<<32,32>>>(gpu_array, n, gpu_counts);
    CSC(hipMemcpy(counts, gpu_counts, sizeof(int) * 256, hipMemcpyDeviceToHost));
    /*for (int i = 0; i < 256; i++) {
        std::cout << counts[i] << " ";
    }
    std::cout << "\n|\n";
    */
    int pref[256];

    prescan<<<1,128>>>(gpu_pref, gpu_counts, 1);
    CSC(hipMemcpy(pref, gpu_pref, sizeof(int) * 256, hipMemcpyDeviceToHost));
    /*std::cout << "pref:\n";
    for (int i = 0; i < 256; i++) {
        std::cout << pref[i] << " ";
    }
*/
    unsigned char* gpu_out;
    CSC(hipMalloc(&gpu_out, sizeof(unsigned char) * n));
    CSC(hipMemset(gpu_out, 0, sizeof(unsigned char) * n));

    kernel<<<32,32>>>(gpu_pref, gpu_out);

    //std::cout << "\n|\n";
    
    CSC(hipMemcpy(array, gpu_out, sizeof(unsigned char) * n, hipMemcpyDeviceToHost));

    /*std::cerr << "out:\n";
    for (int i = 0; i < 10 && i < n; i++) {
        //printf("%02X ", array[i]);
        std::cerr << array[i] << "| ";
    }*/
    fwrite(array, sizeof(unsigned char), n, stdout);

    /*for (int i = n - 200; i < n; i++) {
        printf("%02X ", array[i]);
    }*/
    return 0;
}