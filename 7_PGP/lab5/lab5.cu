#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <vector>
#include <string>

#include <thrust/extrema.h>
#include <thrust/device_vector.h>

#define CSC(call)  \
do { \
	hipError_t state = call; \
	if (state != hipSuccess) { \
		fprintf(stderr, "ERROR: %s:%d. Message: %s\n", __FILE__,__LINE__,hipGetErrorString(state)); \
		exit(0); \
	} \
} while (0); \

__global__ void scan(int* counts)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offsetx = blockDim.x * gridDim.x;
    

    
}

__global__ void kernel(int* pref, unsigned char* out)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offsetx = blockDim.x * gridDim.x;
    
    for(int i = idx; i < n; i += offsetx) {
        out[atomicAdd(pref + i, -1)] = pref[i];
    }
    
}

__global__ void hist(unsigned char* array, int n, int* out) {
    __shared__ int temp[256];
    
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offsetx = blockDim.x * gridDim.x;

    for(int i = idx; i < n; i += offsetx) {
        atomicAdd(temp + array[i], 1);
    }
    __syncthreads();
    if (idx == 0)
    for(int i = 0; i < 256; i ++) {
        atomicAdd(out + i, *(temp + i));
    }
}

int main() {
    int n;
    //std::cin >> std::hex >> n;
    std::cin >> n;
    unsigned char* array = (unsigned char*)malloc(sizeof(unsigned char) * n); //uchar
    for(int i = 0; i < n; i++) {
        //std::cin >> std::hex >> array[i];
        std::cin >> array[i];
    }
    unsigned char* gpu_array;
    CSC(hipMalloc(&gpu_array, sizeof(unsigned char) * n));
    CSC(hipMemcpy(gpu_array, array, sizeof(unsigned char) * n, hipMemcpyHostToDevice));
    
    int* counts = (int*)malloc(sizeof(int) * 256);
    for (int i = 0; i < 256; i++) {
        counts[i] = 0;
    }
    int* gpu_counts;
    CSC(hipMalloc(&gpu_counts, sizeof(int) * 256));
    CSC(hipMemcpy(gpu_counts, counts, sizeof(int) * 256, hipMemcpyHostToDevice));
    
    hist<<<32,32>>>(gpu_array, n, gpu_counts);
    CSC(hipMemcpy(counts, gpu_counts, sizeof(int) * 256, hipMemcpyDeviceToHost));
    std::cout << "\n";
    std::cout << "\n";
    for(int i = 0; i < 256; i++) {
        std::cout << counts[i] << " ";
    }
    std::cout << "\n";

    scan<<<32,32>>>(gpu_counts);
    
    kernel<<<32,32>>>(gpu_counts, gpu_array);
    CSC(hipMemcpy(array, gpu_array, sizeof(unsigned char) * n, hipMemcpyDeviceToHost));

    for(int i = 0; i < n; i++) {
        //std::cout<< std::hex << array[i] << " ";
        std::cout << array[i] << " ";
    }
    return 0;
}