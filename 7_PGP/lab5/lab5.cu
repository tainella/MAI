#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <vector>
#include <string>

#include <thrust/extrema.h>
#include <thrust/device_vector.h>

#define CSC(call)  \
do { \
	hipError_t state = call; \
	if (state != hipSuccess) { \
		fprintf(stderr, "ERROR: %s:%d. Message: %s\n", __FILE__,__LINE__,hipGetErrorString(state)); \
		exit(0); \
	} \
} while (0); \

__device__ int what_pow(int n) {
    int k = 0;
    while (n > 1) {
        n = n / 2;
        k++;
    }
    return k;
}

__host__ int what_pow_h(int n) {
    int k = 0;
    while (n > 1) {
        n = n / 2;
        k++;
    }
    return k;
}

__device__  int no_conflict_offset(int n, int banks) {
    return ((n) >> banks + (n) >> (2 * what_pow(banks)));
}

__global__ void gpu_add_block_sums(int* out, int* const in, int* block_sums)
{ 
    int d_block_sum_val = block_sums[blockIdx.x];
    int cpy_idx = 2 * blockIdx.x * blockDim.x + threadIdx.x;
    if (cpy_idx < 257)
    {
        out[cpy_idx] = in[cpy_idx] + d_block_sum_val;
        if (cpy_idx + blockDim.x < 257)
            out[cpy_idx + blockDim.x] = in[cpy_idx + blockDim.x] + d_block_sum_val;
    }
}

__global__ void prescan(int* d_out, int* d_in, int* block_sums, int shmem_size, int blocks, int max_block_size) {
    __shared__ int temp[257];

    int idx = threadIdx.x;
    int ai = idx;
    int bi = idx + blockDim.x;

    temp[idx] = 0;
    temp[idx + blockDim.x] = 0;
    temp[idx + blockDim.x + (blockDim.x >> what_pow(blocks))] = 0;
    
    __syncthreads();

    int cpy_idx = max_block_size * blockIdx.x + threadIdx.x;
    if (cpy_idx < 257)
    {
        temp[ai + no_conflict_offset(ai, blocks)] = d_in[cpy_idx];
        if (cpy_idx + blockDim.x < 257)
            temp[bi + no_conflict_offset(bi, blocks)] = d_in[cpy_idx + blockDim.x];
    }

    int offset = 1;
    for (int d = max_block_size >> 1; d > 0; d >>= 1)
    {
        __syncthreads();

        if (idx < d)
        {
            int ai = offset * ((idx << 1) + 1) - 1;
            int bi = offset * ((idx << 1) + 2) - 1;
            ai += no_conflict_offset(ai, blocks);
            bi += no_conflict_offset(bi, blocks);

            temp[bi] += temp[ai];
        }
        offset <<= 1;
    }
    
    if (idx == 0) 
    { 
        block_sums[blockIdx.x] = temp[max_block_size - 1 + no_conflict_offset(max_block_size - 1, blocks)];
        temp[max_block_size - 1 + no_conflict_offset(max_block_size - 1, blocks)] = 0;
    }
    for (int d = 1; d < max_block_size; d <<= 1)
    {
        offset >>= 1;
        __syncthreads();

        if (idx < d)
        {
            int ai = offset * ((idx << 1) + 1) - 1;
            int bi = offset * ((idx << 1) + 2) - 1;
            ai += no_conflict_offset(ai, blocks);
            bi += no_conflict_offset(bi, blocks);

            unsigned int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();
    if (cpy_idx < 257)
    {
        d_out[cpy_idx] = temp[ai + no_conflict_offset(ai, blocks)];
        if (cpy_idx + blockDim.x < 257)
            d_out[cpy_idx + blockDim.x] = temp[bi + no_conflict_offset(bi, blocks)];
    }
}
 
void scan(int* counts, int* out, int blocks, int max_block_size) {
    CSC(hipMemset(out, 0, 257 * sizeof(int)));
    int block_size = max_block_size / 2;
    int grid_size = 257 / max_block_size;
    
    if (257 % max_block_size != 0) {
        grid_size += 1;
    }
    int shmem_size = max_block_size + ((max_block_size) >> what_pow_h(blocks)); //к примеру 5
    int* block_sums;
    CSC(hipMalloc(&block_sums, sizeof(int) * grid_size));
    CSC(hipMemset(block_sums, 0, sizeof(int) * grid_size));

    prescan<<<grid_size, block_size, sizeof(int) * shmem_size>>>(out, counts, block_sums, shmem_size, blocks, max_block_size);

    if (grid_size <= max_block_size)
    {
        int* d_dummy_blocks_sums;
        CSC(hipMalloc(&d_dummy_blocks_sums, sizeof(int)));
        CSC(hipMemset(d_dummy_blocks_sums, 0, sizeof(int)));

        prescan<<<1, block_size, sizeof(int) * shmem_size>>>(block_sums, block_sums, d_dummy_blocks_sums, grid_size, shmem_size, max_block_size);
        
        CSC(hipFree(d_dummy_blocks_sums));
    }
    else
    {
        int* d_in_block_sums;
        CSC(hipMalloc(&d_in_block_sums, sizeof(int) * grid_size));
        CSC(hipMemcpy(d_in_block_sums, block_sums, sizeof(int) * grid_size, hipMemcpyDeviceToDevice));

        scan(block_sums, d_in_block_sums, blocks, grid_size);

        CSC(hipFree(d_in_block_sums));
    }
    gpu_add_block_sums<<<grid_size, block_size>>>(out, out, block_sums);
    CSC(hipFree(block_sums));
}

__global__ void kernel(int* pref, unsigned char* out, int n, unsigned char* array)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offsetx = blockDim.x * gridDim.x;

    for(int i = n-1-idx; i >= 0; i -= offsetx){
        out[atomicAdd(pref + array[i]+1, -1)-1] = array[i];
    }
    
}


__global__ void hist(unsigned char* array, int n, int* out) {
    __shared__ int temp[257];
    
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offsetx = blockDim.x * gridDim.x;

    for(int i = idx; i < n; i += offsetx) {
        atomicAdd(temp + array[i], 1);
    }
    __syncthreads();
    if (idx == 0)
    for(int i = 0; i < 257; i ++) {
        atomicAdd(out + i, *(temp + i));
    }
}

int main() {
    int n;
    fread(&n, sizeof(int), 1, stdin);
    unsigned char* array = (unsigned char*)malloc(sizeof(unsigned char) * n); //uchar
    fread(array, sizeof(unsigned char), n, stdin);
    unsigned char* gpu_array;
    CSC(hipMalloc(&gpu_array, sizeof(unsigned char) * n));
    CSC(hipMemcpy(gpu_array, array, sizeof(unsigned char) * n, hipMemcpyHostToDevice));
    
    int* counts = (int*)malloc(sizeof(int) * 257);
    for (int i = 0; i < 257; i++) {
        counts[i] = 0;
    }
    int* gpu_counts;
    CSC(hipMalloc(&gpu_counts, sizeof(int) * 257));
    CSC(hipMemcpy(gpu_counts, counts, sizeof(int) * 257, hipMemcpyHostToDevice));
    
    hist<<<32,32>>>(gpu_array, n, gpu_counts);
    CSC(hipMemcpy(counts, gpu_counts, sizeof(int) * 257, hipMemcpyDeviceToHost));

    int* gpu_pref;
    CSC(hipMalloc(&gpu_pref, sizeof(int) * 257));
    scan(gpu_counts, gpu_pref, 32, 32);
    
    unsigned char* gpu_out;
    CSC(hipMalloc(&gpu_out, sizeof(unsigned char) * n));
    kernel<<<32,32>>>(gpu_pref, gpu_out, n, gpu_array);
    CSC(hipMemcpy(array, gpu_out, sizeof(unsigned char) * n, hipMemcpyDeviceToHost));
    fwrite(array, sizeof(unsigned char), n, stdout);
    return 0;
}