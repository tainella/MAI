#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <vector>
#include <string>

#include <thrust/extrema.h>
#include <thrust/device_vector.h>

#define THREAD_NUM 1024

#define CSC(call)  \
do { \
	hipError_t state = call; \
	if (state != hipSuccess) { \
		fprintf(stderr, "ERROR: %s:%d. Message: %s\n", __FILE__,__LINE__,hipGetErrorString(state)); \
		exit(0); \
	} \
} while (0); \

__device__ int what_pow(int n) {
    int k = 0;
    while (n > 1) {
        n = n / 2;
        k++;
    }
    return k;
}

__host__ int what_pow_h(int n) {
    int k = 0;
    while (n > 1) {
        n = n / 2;
        k++;
    }
    return k;
}

__device__  int no_conflict_offset(int n, int banks) {
    return ((n) >> banks + (n) >> (2 * what_pow(banks)));
}

__global__ void prescan(int* d_out, const int* d_in, int blocks) {
    int sum;
    int max_block_size = 256; //128
    __shared__ int temp[256];

    int idx = threadIdx.x;
    int ai = idx;

    temp[idx] = 0;
    temp[idx + blockDim.x] = 0;
    
    __syncthreads();

    int cpy_idx =  max_block_size * blockIdx.x + threadIdx.x;

    if (cpy_idx < 256)
    {
        //temp[ai + no_conflict_offset(ai, blocks)] = d_in[cpy_idx];
        temp[ai] = d_in[cpy_idx];
    }

    int offset = 1;
    for (int d = max_block_size >> 1; d > 0; d >>= 1)
    {
        __syncthreads();

        if (idx < d)
        {
            int ai = offset * ((idx << 1) + 1) - 1;
            int bi = offset * ((idx << 1) + 2) - 1;
            //ai += no_conflict_offset(ai, blocks);
            //bi += no_conflict_offset(bi, blocks);
           // assert(bi < 256);
            temp[bi] += temp[ai];
        }
        offset <<= 1;
    }
    if (idx == 0) { 
        sum = temp[255];
        temp[255] = 0;
    }
    //обратный ход
    for (int d = 1; d < max_block_size; d <<= 1) {
        offset >>= 1;
        __syncthreads();

        if (idx < d)
        {
            int i1 = offset * ((idx << 1) + 1) - 1;
            int i2 = offset * ((idx << 1) + 2) - 1;
            //i1 += no_conflict_offset(i1, blocks);
            //i2 += no_conflict_offset(i2, blocks);

            int t = temp[i1];
            temp[i1] = temp[i2];
            temp[i2] += t;
        }
    }
    __syncthreads();
    if (cpy_idx == 0) { //переделать из исключающего в включающий
        for (int j = 1; j < 256; j++) {
            d_out[j-1] = temp[j];
        }
         d_out[255] = sum;
         /*for (int j = 0; j < 256; j++) {
            printf("%d ", temp[j]);
         }
         printf("\n|||||||||\n");*/
    }
}

/*
__global__ void kernel(int* pref, unsigned char* out) {
  int prev = 0; 
  for (int i = blockIdx.x; i < 256; i += gridDim.x) {
    if (i > 0) {
      prev = pref[i - 1];
    }

    if (i == 2) 

    for (int j = pref[i] - 1 - threadIdx.x; j >= prev ; j -= blockDim.x) {
      out[j] = i;
    }
  }
}
*/

__global__ void kernel(int* pref, unsigned char* out){
    int idx = blockDim.x * blockIdx.x +  threadIdx.x;
    int step = blockDim.x * gridDim.x;

    for(int tid = idx; tid < 256; tid += step){
        int low = tid ? pref[tid-1] : 0;

        for(int i = pref[tid] - 1; i >= low; --i){
            out[i] = tid-1;
        }
    }
}

__global__ void hist(unsigned char* array, int n, int* out) {
    __shared__ int temp[257];
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offsetx = blockDim.x * gridDim.x;

    for(int i = idx; i < n; i += offsetx) {
        atomicAdd(temp + array[i], 1);
    }
    __syncthreads();
    if (idx == 0)
    for(int i = 0; i < 257; i++) {
        atomicAdd(out + i, *(temp + i));
    }
}

int main() {
    int n;
    fread(&n, sizeof(int), 1, stdin);
    unsigned char* array = (unsigned char*)malloc(sizeof(unsigned char) * n); //uchar
    if (fread(array, sizeof(unsigned char), n, stdin) != n) {
        std::cout << "не считал\n";
    };
    /*for (int i = n - 200; i < n; i++) {
        printf("%02X ", array[i]);
    }*/
    unsigned char* gpu_array;
    CSC(hipMalloc(&gpu_array, sizeof(unsigned char) * n));
    CSC(hipMemcpy(gpu_array, array, sizeof(unsigned char) * n, hipMemcpyHostToDevice));
    
    int* counts = (int*)malloc(sizeof(int) * 256);
    for (int i = 0; i < 256; i++) {
        counts[i] = 0;
    }
    int* gpu_counts;
    CSC(hipMalloc(&gpu_counts, sizeof(int) * 256));
    CSC(hipMemcpy(gpu_counts, counts, sizeof(int) * 256, hipMemcpyHostToDevice));

    int* gpu_pref;
    CSC(hipMalloc(&gpu_pref, sizeof(int) * 256));
    CSC(hipMemset(gpu_pref, 0, sizeof(int) * 256));

    hist<<<32,32>>>(gpu_array, n, gpu_counts);
    CSC(hipMemcpy(counts, gpu_counts, sizeof(int) * 256, hipMemcpyDeviceToHost));
    /*for (int i = 0; i < 256; i++) {
        std::cout << counts[i] << " ";
    }
    std::cout << "\n|\n";
    */
    int pref[256];

    prescan<<<1,128>>>(gpu_pref, gpu_counts, 1);
    CSC(hipMemcpy(pref, gpu_pref, sizeof(int) * 256, hipMemcpyDeviceToHost));
    /*std::cout << "pref:\n";
    for (int i = 0; i < 256; i++) {
        std::cout << pref[i] << " ";
    }
    */

    unsigned char* gpu_out;
    CSC(hipMalloc(&gpu_out, sizeof(unsigned char) * n));
    CSC(hipMemset(gpu_out, 0, sizeof(unsigned char) * n));

    kernel<<<32,32>>>(gpu_pref, gpu_out);

    //std::cout << "\n|\n";
    CSC(hipMemcpy(array, gpu_out, sizeof(unsigned char) * n, hipMemcpyDeviceToHost));
    fwrite(array, sizeof(unsigned char), n, stdout);
    /*for (int i = n - 200; i < n; i++) {
        printf("%02X ", array[i]);
    }*/
    return 0;
}