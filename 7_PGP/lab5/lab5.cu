#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <vector>
#include <string>

#include <thrust/extrema.h>
#include <thrust/device_vector.h>

#define CSC(call)  \
do { \
	hipError_t state = call; \
	if (state != hipSuccess) { \
		fprintf(stderr, "ERROR: %s:%d. Message: %s\n", __FILE__,__LINE__,hipGetErrorString(state)); \
		exit(0); \
	} \
} while (0); \


__global__ void scan(int* counts, int* out)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offsetx = blockDim.x * gridDim.x;
    
    __shared__ unsigned int temp[];
    
    
    __syncthreads();
    
}

__global__ void kernel(int* pref, unsigned char* out, int n, unsigned char* array)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offsetx = blockDim.x * gridDim.x;
    
    for(int i = idx; i < n; i += offsetx) {
        out[atomicAdd(pref + array[i], -1)] = array[i];
    }
    
}

__global__ void hist(unsigned char* array, int n, int* out) {
    __shared__ int temp[256];
    
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offsetx = blockDim.x * gridDim.x;

    for(int i = idx; i < n; i += offsetx) {
        atomicAdd(temp + array[i], 1);
    }
    __syncthreads();
    if (idx == 0)
    for(int i = 0; i < 256; i ++) {
        atomicAdd(out + i, *(temp + i));
    }
}

int main() {
    int n;
    //std::cin >> std::hex >> n;
    std::cin >> n;
    unsigned char* array = (unsigned char*)malloc(sizeof(unsigned char) * n); //uchar
    for(int i = 0; i < n; i++) {
        //std::cin >> std::hex >> array[i];
        std::cin >> array[i];
    }
    unsigned char* gpu_array;
    CSC(hipMalloc(&gpu_array, sizeof(unsigned char) * n));
    CSC(hipMemcpy(gpu_array, array, sizeof(unsigned char) * n, hipMemcpyHostToDevice));
    
    int* counts = (int*)malloc(sizeof(int) * 256);
    for (int i = 0; i < 256; i++) {
        counts[i] = 0;
    }
    int* gpu_counts;
    CSC(hipMalloc(&gpu_counts, sizeof(int) * 256));
    CSC(hipMemcpy(gpu_counts, counts, sizeof(int) * 256, hipMemcpyHostToDevice));
    
    hist<<<32,32>>>(gpu_array, n, gpu_counts);
    CSC(hipMemcpy(counts, gpu_counts, sizeof(int) * 256, hipMemcpyDeviceToHost));
    std::cout << "\n";
    std::cout << "\n";
    for(int i = 0; i < 256; i++) {
        std::cout << counts[i] << " ";
    }
    std::cout << "\n";

    int* gpu_pref;
    CSC(hipMalloc(&gpu_pref, sizeof(int) * 256));
    scan<<<32,32>>>(gpu_counts, gpu_pref);
    
    unsigned char* gpu_out;
    CSC(hipMalloc(&gpu_out, sizeof(unsigned char) * n));
    kernel<<<32,32>>>(gpu_pref, gpu_out, n, gpu_array);
    CSC(hipMemcpy(array, gpu_out, sizeof(unsigned char) * n, hipMemcpyDeviceToHost));

    for(int i = 0; i < n; i++) {
        //std::cout<< std::hex << array[i] << " ";
        std::cout << array[i] << " ";
    }
    return 0;
}