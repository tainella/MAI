#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <vector>
#include <string>

#include <thrust/extrema.h>
#include <thrust/device_vector.h>

#define CSC(call)  \
do { \
	hipError_t state = call; \
	if (state != hipSuccess) { \
		fprintf(stderr, "ERROR: %s:%d. Message: %s\n", __FILE__,__LINE__,hipGetErrorString(state)); \
		exit(0); \
	} \
} while (0); \

__global__ void kernel()
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offsetx = blockDim.x * gridDim.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
	int offsety = blockDim.y * gridDim.y;
    
}

__global__ void hist(unsigned char* array, int n, int* out) {
    __shared__ int temp[256];
    
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offsetx = blockDim.x * gridDim.x;
}

int main() {
    int n;
    std::cin >> std::hex >> n;
    unsigned char* array = (unsigned char*)malloc(sizeof(unsigned char) * n); //uchar
    for(int i = 0; i < n; i++) {
        std::cin >> std::hex >> array[i];
    }
    double* gpu_array;
    CSC(hipMalloc(&gpu_array, sizeof(unsigned char) * n));
    CSC(hipMemcpy(gpu_array, array, sizeof(unsigned char) * n, hipMemcpyHostToDevice));
    
    int* counts = (int*)malloc(sizeof(int) * 256);
    for (int i = 0; i < 256; i++) {
        counts[i] = 0;
    }
    double* gpu_counts;
    CSC(hipMalloc(&gpu_counts, sizeof(int) * 256));
    CSC(hipMemcpy(gpu_counts, counts, sizeof(int) * 256, hipMemcpyHostToDevice));
    
    hist<<<32,32>>>(gpu_array, n, gpu_counts);
    

    
    for(int i = 0; i < n; i++) {
        std::cout<< std::hex << array[i] << " ";
    }
    return 0;
}
