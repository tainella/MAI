
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <vector>
#include <string>

#define CSC(call)  \
do { \
	hipError_t state = call; \
	if (state != hipSuccess) { \
		fprintf(stderr, "ERROR: %s:%d. Message: %s\n", __FILE__,__LINE__,hipGetErrorString(state)); \
		exit(0); \
	} \
} while (0); \

typedef struct pair {
    double x;
    double y;
    double z;
} pair;

__device__ double dist(double x1, double y1, double z1, double x2, double y2, double z2) {
    return sqrt((x1-x2)*(x1-x2) + (y1-y2)*(y1-y2) + (z1-z2)*(z1-z2));
}

__constant__ pair u[32];

__global__ void kernel(uchar4 *data, int w, int h, int n) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;
	int x, y;
	double mas[32];
    for (y = idy; y < h; y += offsety) {
        for(x = idx; x < w; x += offsetx) {
            double4 p = make_double4(data[y * w + x].x, data[y * w + x].y, data[y * w + x].z, 0);//int4
            for (int j = 0; j < n; j++) {
                 mas[j] = dist(u[j].x, u[j].y, u[j].z, p.x, p.y, p.z); //заполняем массив расстояний            
            }
            //поиск минимума
            double MIN = mas[0];
            int min_i = 0;
            for (int j = 1; j < n; j++) {
                if (mas[j] < MIN) {
                    MIN = mas[j];
                    min_i = j; 
                }
            }
            data[y*w+x].w = min_i;
            //printf("%d ", min_i);
        }
    }         
}

int main() {
	std::string inputFile;
	std::string outputFile;
	int n, w, h;
    int x, y;

	std::cin >> inputFile >> outputFile;
	scanf("%d", &n);
    pair centre[n];
    pair centre_new[n];

	FILE* fp = fopen(inputFile.c_str(), "rb");
	
	fread(&w, sizeof(int), 1, fp);
	fread(&h, sizeof(int), 1, fp);
	uchar4* data = (uchar4*)malloc(sizeof(uchar4) * w * h);
	fread(data, sizeof(uchar4), w * h, fp);
	fclose(fp);

    for(int i = 0; i < n; i++) {
        scanf("%d %d", &x, &y);
        double3 g = make_double3(data[y*w+x].x, data[y*w+x].y, data[y*w+x].z);
        centre[i].x = g.x;
        centre[i].y = g.y;
        centre[i].z = g.z;
    }

    uchar4 *dev_data;
    CSC(hipMalloc(&dev_data, sizeof(uchar4) * h * w));
    CSC(hipMemcpy(dev_data, data, sizeof(uchar4) * h * w, hipMemcpyHostToDevice));

    bool flag = true;
    while(flag) {
        CSC(hipMemcpyToSymbol(HIP_SYMBOL(u), centre, sizeof(pair) * n));
        kernel<<<dim3(16, 16), dim3(16, 16)>>>(dev_data, w, h, n); 
        //printf("\n");
        CSC(hipMemcpy(data, dev_data, sizeof(uchar4) * h * w, hipMemcpyDeviceToHost));
       
        //обновление центра кластеров
        int k[n];
        double sum[n][3];
        for(y = 0; y < n; y++) {
            k[y] = 0;
            sum[y][0] = 0;
            sum[y][1] = 0;
            sum[y][2] = 0;
        }
        double4 t;
        for(y = 0; y < h; y++) {
            for(x = 0; x < w; x++) {
                t = make_double4(data[y*w+x].x, data[y*w+x].y, data[y*w+x].z, data[y*w+x].w);
                //printf("%d ", t);
                k[int(t.w)] += 1;
                sum[int(t.w)][0] += t.x;
                sum[int(t.w)][1] += t.y;
                sum[int(t.w)][2] += t.z;
            }
        }
        //printf("\n");
        for(int r = 0; r < n; r++) {
            if (k[r] > 0) {
                centre_new[r].x = sum[r][0] / k[r];
                centre_new[r].y = sum[r][1] / k[r];
                centre_new[r].z = sum[r][2] / k[r];
            }
        }
        
        //условие прекращения
        for(int i = 0; i < n; i++) {
            if (!(centre[i].x == centre_new[i].x && centre[i].y == centre_new[i].y && centre[i].z == centre_new[i].z)) {
                flag = false; //были не одинаковые
                break;
            }  
        }
        if (flag == false) {
            flag = true; //продолжаем обработку
            for(int i = 0; i < n; i++) { 
                centre[i].x = centre_new[i].x;
                centre[i].y = centre_new[i].y;
                centre[i].z = centre_new[i].z;
            }
        }
        else {
            flag = false; //заканчиваем, выход из цикла
        }
    }

    fp = fopen(outputFile.c_str(), "wb");
	fwrite(&w, sizeof(int), 1, fp);
	fwrite(&h, sizeof(int), 1, fp);
	fwrite(data, sizeof(uchar4), w * h, fp);
	fclose(fp);

	free(data);
    CSC(hipFree(dev_data));
	return 0;
}