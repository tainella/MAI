#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <fstream>
#include <algorithm>
#include <stdlib.h>
#include <iomanip>
#include <string.h>
#include <thrust/extrema.h>
#include <thrust/device_vector.h>
#include "mpi.h"

// размер флота 12 + разделитель + минус = 14

#define next_ijk(i, j, k, step) { \
    i += step;  \
    while(i > n_x){ \
        i -= n_x; \
        j++; \
    } \
    while(j > n_y){ \
        j -= n_y; \
        k++; \
    } \
} \

#define next_ij(i, j, step) { \
    i += step;  \
    while(i > n_x){ \
	i -= n_x; \
	j++; \
} \
} \

#define next_ik(i, k, step) { \
    i += step;  \
    while(i > n_x){ \
        i -= n_x; \
        k++; \
    } \
} \

#define next_jk(j, k, step) { \
    j += step;  \
    while(j > n_y){ \
        j -= n_y; \
        k++; \
    } \
} \

#define idx(i, j, k) ((i) + size_x*((j) + (k)*size_y))

__host__ __device__ double u_next(double ux0, double ux1, double uy0, double uy1, double uz0, double uz1, double h2x, double h2y, double h2z) {
    double ans = (ux0 + ux1) * h2x;
    ans += (uy0 + uy1) * h2y; 
    ans += (uz0 + uz1) * h2z;
    return ans;
}

__host__ __device__ double max_determine(double val1, double val2, double curr_max){
    double diff = val1 - val2;
    diff = diff < 0.0 ? -diff : diff;

    return diff > curr_max ? diff : curr_max;
}

__global__ void import_x(double* inner_buff, double* edge_buff1, double* edge_buff2, int n_x, int n_y, int n_z){
    int size_x = n_x + 2;
    int size_y = n_y + 2;
    int num_threads = blockDim.x * gridDim.x;
    int thread_idx = blockDim.x * blockIdx.x +  threadIdx.x;

    int j = 1;
    int k = 1;
    next_jk(j, k, thread_idx);

    while(k <= n_z) {
        inner_buff[idx(0, j, k)] = edge_buff1[j + k * size_y];
        inner_buff[idx(n_x + 1, j, k)] = edge_buff2[j + k * size_y];
        next_jk(j, k, num_threads);
    }
}

__global__ void import_y(double* inner_buff, double* edge_buff1, double* edge_buff2, int n_x, int n_y, int n_z){
    int size_x = n_x + 2;
    int size_y = n_y + 2;
    int num_threads = blockDim.x * gridDim.x;
    int thread_idx = blockDim.x * blockIdx.x +  threadIdx.x;

    int i = 1;
    int k = 1;
    next_ik(i, k, thread_idx);

    while(k <= n_z) {
        inner_buff[idx(i, 0, k)] = edge_buff1[i + k * size_x];
        inner_buff[idx(i, n_y + 1, k)] = edge_buff2[i + k * size_x];
        next_ik(i, k, num_threads);
    }
}

__global__ void import_z(double* inner_buff, double* edge_buff1, double* edge_buff2, int n_x, int n_y, int n_z) {
    int size_x = n_x + 2;
    int size_y = n_y + 2;
    int num_threads = blockDim.x * gridDim.x;
    int thread_idx = blockDim.x * blockIdx.x +  threadIdx.x;

    int i = 1;
    int j = 1;
    next_ij(i, j, thread_idx);

    while(j <= n_y) {
        inner_buff[idx(i, j, 0)] = edge_buff1[i + j * size_x];
        inner_buff[idx(i, j, n_z + 1)] = edge_buff2[i + j * size_x];
        next_ij(i, j, num_threads);
    }
}

__global__ void export_x(double* inner_buff, double* edge_buff1, double* edge_buff2, int n_x, int n_y, int n_z) {
    int size_x = n_x + 2;
    int size_y = n_y + 2;
    int num_threads = blockDim.x * gridDim.x;
    int thread_idx = blockDim.x * blockIdx.x +  threadIdx.x;

    int j = 1;
    int k = 1;
    next_jk(j, k, thread_idx);

    while(k <= n_z){
        edge_buff1[j + k * size_y] = inner_buff[idx(1, j, k)];
        edge_buff2[j + k * size_y] = inner_buff[idx(n_x, j, k)];
        next_jk(j, k, num_threads);
    }
}

__global__ void export_y(double* inner_buff, double* edge_buff1, double* edge_buff2, int n_x, int n_y, int n_z) {
    int size_x = n_x + 2;
    int size_y = n_y + 2;
    int num_threads = blockDim.x * gridDim.x;
    int thread_idx = blockDim.x * blockIdx.x +  threadIdx.x;

    int i = 1;
    int k = 1;
    next_ik(i, k, thread_idx);

    while(k <= n_z) {
        edge_buff1[i + k * size_x] = inner_buff[idx(i, 1, k)];
        edge_buff2[i + k * size_x] = inner_buff[idx(i, n_y, k)];
        next_ik(i, k, num_threads);
    }
}

__global__ void export_z(double* inner_buff, double* edge_buff1, double* edge_buff2, int n_x, int n_y, int n_z){
    int size_x = n_x + 2;
    int size_y = n_y + 2;
    int num_threads = blockDim.x * gridDim.x;
    int thread_idx = blockDim.x * blockIdx.x +  threadIdx.x;

    int i = 1;
    int j = 1;
    next_ij(i, j, thread_idx);

    while(j <= n_y) {
        edge_buff1[i + j * size_x] = inner_buff[idx(i, j, 1)];
        edge_buff2[i + j * size_x] = inner_buff[idx(i, j, n_z)];
        next_ij(i, j, num_threads);
    }
}

__global__ void new_grid(double* buffer1, double* buffer0, double* max_values, int n_x, int n_y, int n_z, double h2x, double h2y, double h2z) {
    int size_x = n_x + 2;
    int size_y = n_y + 2;

    int num_threads = blockDim.x * gridDim.x;
    int thread_idx = blockDim.x * blockIdx.x +  threadIdx.x;

    int i = 1;
    int j = 1;
    int k = 1;
    next_ijk(i, j, k, thread_idx);
    double temp;
    max_values[thread_idx] = 0.0;

    while(k <= n_z){
    temp = u_next(buffer0[idx(i - 1, j, k)], buffer0[idx(i + 1, j, k)], buffer0[idx(i, j - 1, k)], buffer0[idx(i, j + 1, k)], buffer0[idx(i, j, k - 1)], buffer0[idx(i, j, k + 1)], h2x, h2y, h2z);
        max_values[thread_idx] = max_determine(buffer0[idx(i, j, k)], temp, max_values[thread_idx]);
        buffer1[idx(i, j, k)] = temp;
        next_ijk(i, j, k, num_threads);
    }
}

int is_main(int worker) {
    return worker ? 0 : 1;
} 

void recv_waiting(MPI_Request* in, MPI_Request* out) {
    MPI_Status temp;
    MPI_Wait(in, &temp);
    MPI_Wait(out, &temp);
}

void print_line(std::ostream& os, double* line, int size) {
    for(int i = 0; i < size; ++i){
        os << line[i] << " ";
    }
}

void edges_exchange(double** edge_buff_in, double** edge_buff_out, int* dir_edge_sizes, int* coords, int* dimens, int* neighb, MPI_Comm grid_comm) {
    MPI_Request in[6];
    MPI_Request out[6];

    for(int dir = 0; dir < 3; dir++) {
        int dir_x_2 = dir << 1;
        if(coords[dir]) {
            MPI_Isend(edge_buff_out[dir_x_2], dir_edge_sizes[dir], MPI_DOUBLE, neighb[dir_x_2], 0, grid_comm, &out[dir_x_2]);
            MPI_Irecv(edge_buff_in[dir_x_2], dir_edge_sizes[dir], MPI_DOUBLE, neighb[dir_x_2], 0, grid_comm, &in[dir_x_2]);
        }
        if (coords[dir] < dimens[dir] - 1) {
            MPI_Isend(edge_buff_out[dir_x_2 + 1], dir_edge_sizes[dir], MPI_DOUBLE, neighb[dir_x_2 + 1], 0, grid_comm, &out[dir_x_2 + 1]);
            MPI_Irecv(edge_buff_in[dir_x_2 + 1], dir_edge_sizes[dir], MPI_DOUBLE, neighb[dir_x_2 + 1], 0, grid_comm, &in[dir_x_2 + 1]);
        }
    }

    for(int dir = 0; dir < 3; ++dir) {
        int orr = dir << 1;
        if(coords[dir] > 0) {
            recv_waiting(&in[orr], &out[orr]);
        }
        if(coords[dir] < dimens[dir] - 1) {
            recv_waiting(&in[orr + 1], &out[orr + 1]);
        }
    }
}

enum orientation{
        left = 0, right,
        front, back,
        down, up,
    };

int main(int argc, char **argv){
    std::ios_base::sync_with_stdio(false);
    std::cin.tie(NULL);

    int main_worker, proc_rank;
    int workers_count = 0;
    int dimens[3], blocks[3];
    double l[3];
    double u[6];
    double u0, eps;
    std::string path;
    int filename_size;

    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &proc_rank);
    main_worker = is_main(proc_rank);

    if(main_worker) {
        std::cin >> dimens[0] >> dimens[1] >> dimens[2] >> blocks[0] >> blocks[1] >> blocks[2];
        std::cin >> path >> eps >> l[0] >> l[1] >> l[2] >> u[down] >> u[up] >> u[left] >> u[right];
        std::cin >> u[front] >> u[back] >> u0;
        filename_size = path.size();
    }

    MPI_Bcast(dimens, 3, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Bcast(blocks, 3, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Bcast(l, 3, MPI_DOUBLE, 0, MPI_COMM_WORLD);
    MPI_Bcast(u, 6, MPI_DOUBLE, 0, MPI_COMM_WORLD);
    MPI_Bcast(&u0, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
    MPI_Bcast(&eps, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);

    MPI_Bcast(&filename_size, 1, MPI_INT, 0, MPI_COMM_WORLD);
    path.resize(filename_size);
    MPI_Bcast((char*) path.c_str(), filename_size, MPI_CHAR, 0, MPI_COMM_WORLD);

    MPI_Comm grid_comm;
    int perod[3];
    int coords[3];
    int neighb[6];

    std::fill_n(perod, 3, 0);

    MPI_Cart_create(MPI_COMM_WORLD, 3, dimens, perod, false, &grid_comm);
    MPI_Comm_rank(grid_comm, &proc_rank);
    MPI_Comm_size(grid_comm, &workers_count); 
    MPI_Cart_coords(grid_comm, proc_rank, 3, coords); 
    main_worker = is_main(proc_rank);

    MPI_Cart_shift(grid_comm, 0, 1, &neighb[left], &neighb[right]); 
    MPI_Cart_shift(grid_comm, 1, 1, &neighb[front], &neighb[back]);
    MPI_Cart_shift(grid_comm, 2, 1, &neighb[down], &neighb[up]);

    int device_count;
    hipGetDeviceCount(&device_count);
    hipSetDevice(proc_rank % device_count);

    double h2x, h2y, h2z; 
    h2x = l[0] / ((double)dimens[0] * blocks[0]);
    h2y = l[1] / ((double)dimens[1] * blocks[1]);
    h2z = l[2] / ((double)dimens[2] * blocks[2]);

    h2x *= h2x;
    h2y *= h2y;
    h2z *= h2z;

    double denuminator = 2.0 * (1.0 / h2x + 1.0 / h2y + 1.0 / h2z);
    h2x = 1.0 / (denuminator * h2x);
    h2y = 1.0 / (denuminator * h2y);
    h2z = 1.0 / (denuminator * h2z);

    int dir_edge_sizes[3];
    int size_x = blocks[0] + 2;
    int size_y = blocks[1] + 2;
    int size_z = blocks[2] + 2;
    dir_edge_sizes[0] = size_y * size_z;
    dir_edge_sizes[1] = size_x * size_z;
    dir_edge_sizes[2] = size_y * size_x;
    
    double* norm_data = new double[workers_count];
    double* h_edge_buff_out[6]; 
    double* h_edge_buff_in[6];
    double* h_buff0;
    double* d_edge_buff_out[6]; 
    double* d_edge_buff_in[6];
    double* d_buff0;
    double* d_buff1;
    double* d_maxvalues;

    int buff_size = size_x * size_y * size_z;
    h_buff0 = new double[buff_size];
    std::fill_n(h_buff0, buff_size, u0);
    hipMalloc((void**) &d_buff0, sizeof(double) * buff_size);
    hipMalloc((void**) &d_buff1, sizeof(double) * buff_size);
    hipMemcpy(d_buff0, h_buff0, sizeof(double) * buff_size, hipMemcpyHostToDevice);
    hipMalloc((void**) &d_maxvalues, sizeof(double) * 16 * 16);

    for(int dir = 0; dir < 3; dir++){
        int dir_x_2 = dir << 1;
        h_edge_buff_out[dir_x_2] = new double[dir_edge_sizes[dir]];
        h_edge_buff_in[dir_x_2] = new double[dir_edge_sizes[dir]];
        h_edge_buff_out[dir_x_2 + 1] = new double[dir_edge_sizes[dir]];
        h_edge_buff_in[dir_x_2 + 1] = new double[dir_edge_sizes[dir]];

        hipMalloc((void**) &d_edge_buff_in[dir_x_2], sizeof(double) * dir_edge_sizes[dir]);
        hipMalloc((void**) &d_edge_buff_in[dir_x_2 + 1], sizeof(double) * dir_edge_sizes[dir]);
        hipMalloc((void**) &d_edge_buff_out[dir_x_2], sizeof(double) * dir_edge_sizes[dir]);
        hipMalloc((void**) &d_edge_buff_out[dir_x_2 + 1], sizeof(double) * dir_edge_sizes[dir]);

        std::fill_n(h_edge_buff_out[dir_x_2], dir_edge_sizes[dir], u0);
        std::fill_n(h_edge_buff_out[dir_x_2 + 1], dir_edge_sizes[dir], u0);

        if(!coords[dir]) {
            std::fill_n(h_edge_buff_in[dir_x_2], dir_edge_sizes[dir], u[dir_x_2]);
        }
        if(coords[dir] == dimens[dir] - 1) {
            std::fill_n(h_edge_buff_in[dir_x_2 + 1], dir_edge_sizes[dir], u[dir_x_2 + 1]);
        }
    }

    double max_diff = 0.0;
    thrust::device_ptr<double> i_ptr = thrust::device_pointer_cast(d_maxvalues);

    do {
        edges_exchange(h_edge_buff_in, h_edge_buff_out, dir_edge_sizes, coords, dimens, neighb, grid_comm);

        for(int orr = left; orr <= up; orr++){
            hipMemcpy(d_edge_buff_in[orr], h_edge_buff_in[orr], sizeof(double) * dir_edge_sizes[orr >> 1], hipMemcpyHostToDevice);
        }

        import_x<<<16, 16>>>(d_buff0, d_edge_buff_in[left], d_edge_buff_in[right], blocks[0], blocks[1], blocks[2]);
        import_y<<<16, 16>>>(d_buff0, d_edge_buff_in[front], d_edge_buff_in[back], blocks[0], blocks[1], blocks[2]);
        import_z<<<16, 16>>>(d_buff0, d_edge_buff_in[down], d_edge_buff_in[up], blocks[0], blocks[1], blocks[2]);
        
        hipDeviceSynchronize();

        new_grid<<<16, 16>>>(d_buff1, d_buff0, d_maxvalues, blocks[0], blocks[1], blocks[2], h2x, h2y, h2z);

        hipDeviceSynchronize();

        export_x<<<16, 16>>>(d_buff1, d_edge_buff_out[left], d_edge_buff_out[right], blocks[0], blocks[1], blocks[2]);
        export_y<<<16, 16>>>(d_buff1, d_edge_buff_out[front], d_edge_buff_out[back], blocks[0], blocks[1], blocks[2]);
        export_z<<<16, 16>>>(d_buff1, d_edge_buff_out[down], d_edge_buff_out[up], blocks[0], blocks[1], blocks[2]);

        hipDeviceSynchronize();

        for(int orr = left; orr <= up; orr++){
            hipMemcpy(h_edge_buff_out[orr], d_edge_buff_out[orr], sizeof(double) * dir_edge_sizes[orr >> 1], hipMemcpyDeviceToHost);
        }

        max_diff = *thrust::max_element(i_ptr, i_ptr + 16 * 16);

        MPI_Allgather(&max_diff, 1, MPI_DOUBLE, norm_data, 1, MPI_DOUBLE, grid_comm);

        max_diff = 0.0;
        for(int i = 0; i < workers_count; i++){
            max_diff = max_diff < norm_data[i] ? norm_data[i] : max_diff;
        }

        double* tmp = d_buff1;
        d_buff1 = d_buff0;
        d_buff0 = tmp;
    } while(max_diff >= eps);

    hipMemcpy(h_buff0, d_buff0, sizeof(double) * buff_size, hipMemcpyDeviceToHost);

    char* write_data = new char[buff_size * 14];
    for(int k = 1; k <= blocks[2]; k++){
        for(int j = 1; j <= blocks[1]; j++){
            int i, len;
            for(i = 1; i < blocks[0]; i++){
                len = sprintf(&write_data[idx(i, j, k) * 14], "%.*e ", 6, h_buff0[idx(i, j, k)]);

                if (len < 14) {
                    write_data[idx(i, j, k) * 14 + len] = ' ';
                }
            }
            len = sprintf(&write_data[idx(i, j, k) * 14], "%.*e\n", 6, h_buff0[idx(i, j, k)]);
            if(len < 14){
                write_data[idx(i, j, k) * 14 + len] = '\n';
            }
        }
    }
    MPI_Datatype float_r;
    MPI_Type_contiguous(14, MPI_CHAR, &float_r); 
    MPI_Type_commit(&float_r); 

    MPI_Datatype local_array, global_array;
    int sizes[3];
    int starts[3];
    int gsizes[3];
    int gstarts[3];

    sizes[0] = size_x;
    sizes[1] = size_y;
    sizes[2] = size_z;
    starts[0] = starts[1] = starts[2] = 1;

    gsizes[0] = blocks[0] * dimens[0];
    gsizes[1] = blocks[1] * dimens[1];
    gsizes[2] = blocks[2] * dimens[2];

    gstarts[0] = blocks[0] * coords[0];
    gstarts[1] = blocks[1] * coords[1];
    gstarts[2] = blocks[2] * coords[2];

    MPI_Type_create_subarray(3, sizes, blocks, starts, MPI_ORDER_FORTRAN, float_r, &local_array);
    MPI_Type_create_subarray(3, gsizes, blocks, gstarts, MPI_ORDER_FORTRAN, float_r, &global_array);
    MPI_Type_commit(&local_array);
    MPI_Type_commit(&global_array);

    MPI_File fh;
    MPI_File_delete(path.c_str(), MPI_INFO_NULL);
    MPI_File_open(grid_comm, path.c_str(), MPI_MODE_CREATE | MPI_MODE_RDWR, MPI_INFO_NULL, &fh);
    MPI_File_set_view(fh, 0, MPI_CHAR, global_array, "native", MPI_INFO_NULL);
    MPI_File_write_all(fh, write_data, 1, local_array, MPI_STATUS_IGNORE);
    MPI_File_close(&fh);
    MPI_Finalize();

    delete[] norm_data;
    delete[] h_buff0;
    delete[] write_data;
    hipFree(d_buff0);
    hipFree(d_buff1);
    hipFree(d_maxvalues);

    for(int dir = 0; dir < 3; dir++){
        int dir_x_2 = dir << 1;
        delete[] h_edge_buff_out[dir_x_2];
        hipFree(d_edge_buff_out[dir_x_2]);
        delete[] h_edge_buff_in[dir_x_2];
        hipFree(d_edge_buff_in[dir_x_2]);
        delete[] h_edge_buff_out[dir_x_2 + 1];
        hipFree(d_edge_buff_out[dir_x_2 + 1]);
        delete[] h_edge_buff_in[dir_x_2 + 1];
        hipFree(d_edge_buff_in[dir_x_2 + 1]);
    }    

    return 0;
}