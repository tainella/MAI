
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define CSC(call)  \
do { \
	hipError_t state = call; \
	if (state != hipSuccess) { \
		fprintf(stderr, "ERROR: %s:%d. Message: %s\n", __FILE__,__LINE__,hipGetErrorString(state)); \
		exit(0); \
	} \
} while (0); \

__global__ void kernel(int* v1, long long n, int a0, int d) {
	long long i = blockDim.x * blockIdx.x + threadIdx.x;
	long long offset = blockDim.x * gridDim.x;

	while (i < n) {
		v1[i] = a0 + d * i;
		i = i + offset;
	}
}

int main() {
	long long n = 1500000000;
	//scanf("%lld", &n);

    int a0 = 1;
    //scanf("%d", &a0);

    int d = 1;
    //scanf("%d", &d);
    //for (n = 4; n <= 256; n = n * 2) {
        int* v1 = (int*)malloc(n * sizeof(int)); //элементы прогрессии 

        int* dev_v1;
        CSC(hipMalloc(&dev_v1, sizeof(int) * n));
        hipEvent_t start, end;
        CSC(hipEventCreate(&start));
        CSC(hipEventCreate(&end));
        CSC(hipEventRecord(start));

        kernel<<<256,256>>>(dev_v1, n, a0, d);

        CSC(hipEventRecord(end));
        CSC(hipEventSynchronize(end));
        float t;
        CSC(hipEventElapsedTime(&t, start, end));
        CSC(hipEventDestroy(start));
        CSC(hipEventDestroy(end));
        printf("time: %.10lf\n", t);

        CSC(hipMemcpy(v1, dev_v1, sizeof(int) * n, hipMemcpyDeviceToHost));
        CSC(hipFree(dev_v1));

        for (long long i = 0; i < n; i++) {
            printf("%d ", v1[i]);
        }
        printf("\n\n");
        free(v1);
        printf("time: %.10lf\n", t);
    //}
	return 0;
}